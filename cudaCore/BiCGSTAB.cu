#include "hip/hip_runtime.h"
#include "BICGSTAB.cuh"
#include<vector>
#include<iostream>

CudaVector::~CudaVector()
{
}

void CudaVector::AllocateData(const Vector& vec)
{
	auto v = vec.generateScalar();
	thrust::copy(v.begin(),v.end(),values.begin());
}

CudaSPVector::~CudaSPVector()
{
}

void CudaSPVector::AllocateData(const Vector& vector)
{
	std::vector<idxType> ii;
	std::vector<Scalar> vv;
	for (int i = 0; i < vector.size(); i++)
	{
		if(vector[i] != 0)
			ii.push_back(i),vv.push_back(vector[i]);
	}
	
}

CudaSPMatrix::~CudaSPMatrix()
{
	// for (int i = 0; i < row; i++)
	// {
	// 	hipFree(matrix[i]);
	// }
	// hipFree(dev_matrix);
	// hipFree(preA);
	// delete[] matrix;
}

void CudaSPMatrix::AllocateData(const SymetrixSparseMatrix& mat)
{
	// matrix = new IndexValue*[row];
	// auto prea = new int[row];
	// hipMalloc((void**)&preA, sizeof(int) * row);
	std::vector<idxType> co(MaxCol * row),ro(row);
	std::vector<Scalar> va(MaxCol * row);

	for (int i = 0; i < row; i++)
	{
		// std::vector<IndexValue> vv;
		for (auto& kv:mat.getRow(i))
		{
			if(kv.second != 0)
			{
				// vv.push_back({(int)kv.first,kv.second});
				co[ro[i] * row + i] = kv.first;
				va[row * ro[i] + i] = kv.second,ro[i]++;
			}
		}
		// prea[i] = vv.size();
		// hipMalloc((void**)&matrix[i], sizeof(IndexValue) * vv.size());
		// hipMemcpy(matrix[i], vv.data(), sizeof(IndexValue) * vv.size(),hipMemcpyHostToDevice);
	}
	colume = {co.begin(),co.end()};
	value = {va.begin(),va.end()};
	// hipMemcpy(preA, prea, sizeof(int) * row,hipMemcpyHostToDevice);
	// delete []prea;
	// hipMalloc((void**)&dev_matrix, sizeof(IndexValue*) * row);
	// hipMemcpy(dev_matrix, matrix, sizeof(IndexValue*) * row,hipMemcpyHostToDevice);
}

__global__ void computeP(Scalar* p,Scalar* r,Scalar* v,int length,Scalar beta,Scalar w)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	p[id] = r[id] + beta * (p[id] - w * v[id]);
}


__global__ void computeP_PCG(Scalar* p,Scalar* z,int length,Scalar beta)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	p[id] = z[id] + beta * p[id];
}

__global__ void MatrixMultVector(Scalar* v1,Scalar* v2,IndexValue** matrix,int* preA,int length)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	v1[id] = 0;
	for (int i = 0; i < preA[id]; i++)
	{
		v1[id] += matrix[id][i].value * v2[matrix[id][i].colid];
	}
}

__global__ void computeS(Scalar* s,Scalar* r,Scalar* v,int length,Scalar alpha)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	s[id] = r[id] - alpha * v[id];
}

__global__ void computeX(Scalar* x,Scalar* p,Scalar* s,int length,Scalar alpha,double w)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	x[id] = x[id] + alpha * p[id] + w * s[id];
}

__global__ void MatrixMultVector_ELL(Scalar* v1,Scalar* v2,idxType* col,Scalar* values,int row,int batch)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	// if(id >= length)
	// 	return;
	id *= batch;
	int len = min(row,id + batch);
	// #pragma unroll
	for(;id < len;id++)
	{
		v1[id] = 0;
		for (int i = 0; i < MaxCol; i++)
		{
			if(values[i * row + id] == 0)
				break;
			v1[id] += values[i * row + id] * v2[col[i * row + id]];
		}
	}
}

__global__ void computeP_CG(Scalar* p,Scalar* r,int length,Scalar beta,int batch)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	// if(id >= length)
	// 	return;
	id *= batch;
	int len = min(length,id + batch);
	// float3* pp = reinterpret_cast<float3*>(p);
	// float3* rr = reinterpret_cast<float3*>(r);
	for (; id < len; id+=1)
	{
		p[id] = r[id] + beta * p[id];
		// pp[id].x = rr[id].x + beta * pp[id].x;
		// pp[id].y = rr[id].y + beta * pp[id].y;
		// pp[id].z = rr[id].z + beta * pp[id].z;
		// float3 rrr = rr[id];
		// pp[id].x = rrr.x + beta * pp[id].x;
		// pp[id].y = rrr.y + beta * pp[id].y;
		// pp[id].z = rrr.z + beta * pp[id].z;
		// p[id + 1] = r[id + 1] + beta * p[id + 1];
		// p[id + 2] = r[id + 2] + beta * p[id + 2];
	}
}

__global__ void computeX_CG(Scalar* x,Scalar* p,int length,Scalar alpha,int batch)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	// if(id >= length)
	// 	return;
	id *= batch;
	int len = min(length,id + batch);
	for (; id < len; id++)
	{
		x[id] = x[id] + alpha * p[id] ;
	}
}

__global__ void computeR_CG(Scalar* r,Scalar* Ap,int length,Scalar alpha,int batch)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	// if(id >= length)
	// 	return;
	id *= batch;
	int len = min(length,id + batch);
	for (; id < len; id++)
	{
		r[id] = r[id] - alpha * Ap[id];
	}
}

// __global__ void computeX_R_CG(Scalar* x,Scalar* p,Scalar* r,Scalar* Ap,int length,double alpha,int batch)
__global__ void computeX_R_CG(Scalar* x,Scalar* p,Scalar* r,Scalar* Ap,int length,Scalar alpha,int batch)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	// if(id >= length)
	// 	return;
	id *= batch;
	int len = min(length,id + batch);
	// float3* xx = reinterpret_cast<float3*>(x);
	// float3* rr = reinterpret_cast<float3*>(r);
	// float3* pp = reinterpret_cast<float3*>(p);
	// float3* AApp = reinterpret_cast<float3*>(Ap);
	for (; id < len; id += 1)
	{
		// float3 xxx = xx[id];
		// float3 rrr = rr[id];
		// float3 ppp = pp[id];
		// float3 AAAppp = AApp[id];

		// xxx.x += alpha * ppp.x;
		// xxx.y += alpha * ppp.y;
		// xxx.z += alpha * ppp.z;

		// rrr.x -= alpha * AAAppp.x;
		// rrr.y -= alpha * AAAppp.y;
		// rrr.z -= alpha * AAAppp.z;

		// xx[id] = xxx;
		// rr[id] = rrr;

		x[id] = x[id] + alpha * p[id];
		r[id] = r[id] - alpha * Ap[id];

		// x[id + 1] = x[id + 1] + alpha * p[id + 1];
		// r[id + 1] = r[id + 1] - alpha * Ap[id + 1];

		// x[id + 2] = x[id + 2] + alpha * p[id + 2];
		// r[id + 2] = r[id + 2] - alpha * Ap[id + 2];
	}
}

__global__ void computeX_PCG(Scalar* x,Scalar* p,int length,Scalar alpha)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	x[id] = x[id] + alpha * p[id] ;
}

__global__ void computeR(Scalar* r,Scalar* s,Scalar* t,int length,Scalar w)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	r[id] = s[id] - w * t[id];
}

__global__ void computeR_PCG(Scalar* r,Scalar* w,int length,Scalar alpha)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	r[id] = r[id] - alpha * w[id];
}

void SolveTriL(const SymetrixSparseMatrix& m,thrust::host_vector<Scalar>& x,const thrust::host_vector<Scalar>& b)
{
    for (int i = 0; i < m.get_row(); i++)
    {
        double rest = b[i];
        for(auto& col:m.getRow(i))
        // for(int i = 0;i < 81;i++)
        {
            // rest -= 88 * x[0];
            if(col.first == i)
                break;
            rest -= col.second * x[col.first];
        }
        x[i] = rest / m.index(i,i);
    }
}

void SolveTriU(const SymetrixSparseMatrix& m,thrust::host_vector<Scalar>& x,const thrust::host_vector<Scalar>& b)
{
    for (int i = m.get_row() - 1; i >= 0; i--)
    {
        double rest = b[i];
        for(auto& col:m.getRow(i))
        // for(int i = 0;i < 81;i++)
        {
            // rest -= 88 * x[0];
            if(col.first == i)
                continue;
            rest -= col.second * x[col.first];
        }
        x[i] = rest / m.index(i,i);
    }
}

void CG(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
	const int batch = 1;
	const int ts = 32 * batch;
	int bs = (A.get_row() / ts * ts == A.get_row()) ?  A.get_row() / ts : A.get_row() / ts + 1;
	// int bs = 32;
	dim3 blockSize(bs);
	dim3 blockSize2(bs / 3);
	dim3 threadSize(ts / batch);
	// int mult = bs * ts;
	// int batch = (A.get_row() / mult * mult == A.get_row()) ?  A.get_row() / mult : A.get_row() / mult + 1;
	Scalar alpha = 0.0,rr0,rr1,beta = 0.0;

	CudaSPMatrix cspm(A.get_row(),A.get_col(),A);

	thrust::device_vector<Scalar> r(b.begin(),b.end()),xx(b.size()),p = r,Ap(b.size()),temp(b.size());

	iter = 0;
	norm = 1000;
	double normb = b.norm1();
	thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
	rr1 = thrust::reduce(thrust::device,temp.begin(),temp.end());

	// std::vector<Scalar> tempp;

	while(iter < limit && norm > tolerance * normb)
	{
		MatrixMultVector_ELL<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&Ap[0]),thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&(cspm.colume[0])),thrust::raw_pointer_cast(&(cspm.value[0])),Ap.size(),batch);
		// tempp = {Ap.begin(),Ap.end()};
		thrust::transform(thrust::device,p.begin(),p.end(),Ap.begin(),temp.begin(),thrust::multiplies<Scalar>());
		alpha = rr1 / thrust::reduce(thrust::device,temp.begin(),temp.end());

		computeX_R_CG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&xx[0]),thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&r[0]),thrust::raw_pointer_cast(&Ap[0]),r.size(),alpha,batch);
		// tempp = {r.begin(),r.end()};

		rr0 = rr1;
		thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
		rr1 = thrust::reduce(thrust::device,temp.begin(),temp.end());
		beta = rr1 / rr0;
		computeP_CG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&r[0]),p.size(),beta,batch);
		// tempp = {p.begin(),p.end()};
		
		iter++;
		norm = std::sqrt(rr1);
	}
	x.setvalues({xx.begin(),xx.end()});
}

void PCG_ICC(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
	const int batch = 1;
	const int ts = 32 * batch;
	int bs = (A.get_row() / ts * ts == A.get_row()) ?  A.get_row() / ts : A.get_row() / ts + 1;
	dim3 blockSize(bs);
	dim3 threadSize(ts);
	Scalar alpha = 0.0,rr0,rr1,beta = 0.0;
	// auto precon = A.ichol().inverse_lowertri();
	auto precon = A.ichol();
	auto preconT = precon.transpose();
	CudaSPMatrix cspm(A.get_row(),A.get_col(),A);
	// CudaSPMatrix prec(precon.get_row(),precon.get_col(),precon);
	// CudaSPMatrix precT(preconT.get_row(),preconT.get_col(),preconT);

	thrust::device_vector<Scalar> r(b.begin(),b.end()),xx(b.size()),p(b.size()),Ap(b.size()),temp(b.size()),
	y(b.size()),z(b.size()),w(b.size()),lastr(b.size());

    auto startT = std::chrono::high_resolution_clock::now();
	std::cout << "start!" << std::endl;

	// MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&y[0]),thrust::raw_pointer_cast(&r[0]),prec.dev_matrix,prec.preA,y.size());
	// MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&z[0]),thrust::raw_pointer_cast(&y[0]),precT.dev_matrix,precT.preA,z.size());
	thrust::host_vector<Scalar> r_host(b.begin(),b.end()),y_host(b.size()),z_host(b.size());
	SolveTriL(precon,y_host,r_host);
	SolveTriU(preconT,z_host,y_host);
	z = {z_host.begin(),z_host.end()};
	p = z;
	std::vector<Scalar> tempp{z.begin(),z.end()};
	// std::vector<Scalar> tempp2{y.begin(),y.end()};
	iter = 0;
	thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
	norm = 1000;
	double normb = b.norm1();

	thrust::transform(thrust::device,r.begin(),r.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
	rr0 = thrust::reduce(thrust::device,temp.begin(),temp.end());

	while(iter < limit && norm > tolerance * normb)
	{
		// MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&w[0]),thrust::raw_pointer_cast(&p[0]),cspm.dev_matrix,cspm.preA,Ap.size());
		MatrixMultVector_ELL<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&Ap[0]),thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&(cspm.colume[0])),thrust::raw_pointer_cast(&(cspm.value[0])),Ap.size(),batch);

		// std::vector<Scalar> tempp{w.begin(),w.end()};
		thrust::transform(thrust::device,p.begin(),p.end(),Ap.begin(),temp.begin(),thrust::multiplies<Scalar>());
		alpha = rr0 / thrust::reduce(thrust::device,temp.begin(),temp.end());

		computeX_PCG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&xx[0]),thrust::raw_pointer_cast(&p[0]),xx.size(),alpha);
		// tempp = {xx.begin(),xx.end()};

		// lastr = r;
		computeR_PCG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&r[0]),thrust::raw_pointer_cast(&Ap[0]),r.size(),alpha);
		// tempp = {r.begin(),r.end()};

		// MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&y[0]),thrust::raw_pointer_cast(&r[0]),prec.dev_matrix,prec.preA,y.size());
		// MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&z[0]),thrust::raw_pointer_cast(&y[0]),precT.dev_matrix,precT.preA,z.size());
		r_host= {r.begin(),r.end()};
		SolveTriL(precon,y_host,r_host);
		SolveTriU(preconT,z_host,y_host);
		z = {z_host.begin(),z_host.end()};
		// tempp = {z.begin(),z.end()};

		// thrust::transform(thrust::device,r.begin(),r.end(),lastr.begin(),temp.begin(),thrust::minus<Scalar>());
		// thrust::transform(thrust::device,temp.begin(),temp.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
		thrust::transform(thrust::device,r.begin(),r.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
		rr1 = thrust::reduce(thrust::device,temp.begin(),temp.end());

		beta = rr1 / rr0;
		// thrust::transform(thrust::device,r.begin(),r.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
		// rr0 = thrust::reduce(thrust::device,temp.begin(),temp.end());
		rr0 = rr1;

		computeP_PCG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&z[0]),p.size(),beta);
		// tempp = {p.begin(),p.end()};

		iter++;
		// thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
		// norm = thrust::reduce(thrust::device,temp.begin(),temp.end());
		norm = std::sqrt(rr1);
		std::cout << iter << " " << norm << std::endl;
	}
	x.setvalues({xx.begin(),xx.end()});
    auto endT = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endT - startT);
    
    printf("SOLVETIME = %.5le \n", static_cast<double>(duration.count()));
}

void PCG_SSOR(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
	int bs = (A.get_row() / 32 * 32 == A.get_row()) ?  A.get_row() / 32 : A.get_row() / 32 + 1;
	dim3 blockSize(bs);
	dim3 threadSize(32);
	Scalar alpha = 0.0,rr0,rr1,beta = 0.0;

	auto precon = A.SSORAI();
	auto preconT = precon.transpose();
	CudaSPMatrix cspm(A.get_row(),A.get_col(),A);
	CudaSPMatrix prec(precon.get_row(),precon.get_col(),precon);
	CudaSPMatrix precT(precon.get_row(),precon.get_col(),preconT);

	thrust::device_vector<Scalar> r(b.begin(),b.end()),xx(b.size()),p(b.size()),Ap(b.size()),temp(b.size()),z(b.size()),w(b.size()),lastr(b.size());

	MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&temp[0]),thrust::raw_pointer_cast(&r[0]),prec.dev_matrix,prec.preA,r.size());
	MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&z[0]),thrust::raw_pointer_cast(&temp[0]),precT.dev_matrix,precT.preA,z.size());
	p = z;
	std::vector<Scalar> tempp{z.begin(),z.end()};
	iter = 0;
	// thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
	norm = 1000;
	double normb = b.norm1();

	thrust::transform(thrust::device,r.begin(),r.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
	rr0 = thrust::reduce(thrust::device,temp.begin(),temp.end());

	while(iter < limit && norm > tolerance * normb)
	{
		MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&w[0]),thrust::raw_pointer_cast(&p[0]),cspm.dev_matrix,cspm.preA,Ap.size());

		// std::vector<Scalar> tempp{w.begin(),w.end()};
		// tempp = {w.begin(),w.end()};
		thrust::transform(thrust::device,p.begin(),p.end(),w.begin(),temp.begin(),thrust::multiplies<Scalar>());
		alpha = rr0 / thrust::reduce(thrust::device,temp.begin(),temp.end());

		computeX_PCG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&xx[0]),thrust::raw_pointer_cast(&p[0]),xx.size(),alpha);
		// tempp = {xx.begin(),xx.end()};

		// lastr = r;
		computeR_PCG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&r[0]),thrust::raw_pointer_cast(&w[0]),r.size(),alpha);
		// tempp = {r.begin(),r.end()};

		MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&temp[0]),thrust::raw_pointer_cast(&r[0]),prec.dev_matrix,prec.preA,r.size());
		MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&z[0]),thrust::raw_pointer_cast(&temp[0]),precT.dev_matrix,precT.preA,z.size());
		// tempp = {z.begin(),z.end()};


		// thrust::transform(thrust::device,r.begin(),r.end(),lastr.begin(),temp.begin(),thrust::minus<Scalar>());
		thrust::transform(thrust::device,r.begin(),r.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
		rr1 = thrust::reduce(thrust::device,temp.begin(),temp.end());

		beta = rr1 / rr0;
		// thrust::transform(thrust::device,r.begin(),r.end(),z.begin(),temp.begin(),thrust::multiplies<Scalar>());
		// rr0 = thrust::reduce(thrust::device,temp.begin(),temp.end());
		rr0 = rr1;

		// tempp = {p.begin(),p.end()};
		computeP_PCG<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&z[0]),p.size(),beta);

		iter++;
		thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
		norm = thrust::reduce(thrust::device,temp.begin(),temp.end());
		norm = std::sqrt(norm);
	}
	x.setvalues({xx.begin(),xx.end()});
}

void BiCGSTAB(const SymetrixSparseMatrix& A,Vector& x,const Vector& b,double tolerance,int limit,int& iter,double& norm)
{
	int bs = (A.get_row() / 32 * 32 == A.get_row()) ?  A.get_row() / 32 : A.get_row() / 32 + 1;
	dim3 blockSize(bs);
	dim3 threadSize(32);
	Scalar rho0,w,alpha,rho1;
	rho0 = w = alpha = 1.0;

	CudaSPMatrix cspm(A.get_row(),A.get_col(),A);
	thrust::device_vector<Scalar> r(b.begin(),b.end()),xx(b.size()),r_hat = r,v(b.size()),p(b.size()),s(b.size()),t(b.size()),temp(b.size());
	iter = 0;
	norm = 1000;
	double normb = b.norm1();
	while(iter < limit && norm > tolerance * normb)
	{
		thrust::transform(thrust::device,r_hat.begin(),r_hat.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
		rho1 = thrust::reduce(thrust::device,temp.begin(),temp.end());
		double beta = rho1 / rho0 * alpha / w;
		rho0 = rho1;

		computeP<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&r[0]),thrust::raw_pointer_cast(&v[0]),p.size(),beta,w);
		MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&v[0]),thrust::raw_pointer_cast(&p[0]),cspm.dev_matrix,cspm.preA,v.size());

		thrust::transform(thrust::device,r_hat.begin(),r_hat.end(),v.begin(),temp.begin(),thrust::multiplies<Scalar>());
		alpha = rho1 / thrust::reduce(thrust::device,temp.begin(),temp.end());

		computeS<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&s[0]),thrust::raw_pointer_cast(&r[0]),thrust::raw_pointer_cast(&v[0]),s.size(),alpha);
		MatrixMultVector<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&t[0]),thrust::raw_pointer_cast(&s[0]),cspm.dev_matrix,cspm.preA,t.size());


		thrust::transform(thrust::device,s.begin(),s.end(),t.begin(),temp.begin(),thrust::multiplies<Scalar>());
		w = thrust::reduce(thrust::device,temp.begin(),temp.end());
		thrust::transform(thrust::device,t.begin(),t.end(),t.begin(),temp.begin(),thrust::multiplies<Scalar>());
		w = w / thrust::reduce(thrust::device,temp.begin(),temp.end());


		// thrust::transform(thrust::device,r_hat.begin(),r_hat.end(),t.begin(),temp.begin(),thrust::multiplies<Scalar>());
		// rho1 = -w * thrust::reduce(thrust::device,temp.begin(),temp.end());
		computeX<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&xx[0]),thrust::raw_pointer_cast(&p[0]),thrust::raw_pointer_cast(&s[0]),xx.size(),alpha,w);
		computeR<<<blockSize,threadSize>>>(thrust::raw_pointer_cast(&r[0]),thrust::raw_pointer_cast(&s[0]),thrust::raw_pointer_cast(&t[0]),r.size(),w);
		iter++;
		thrust::transform(thrust::device,r.begin(),r.end(),r.begin(),temp.begin(),thrust::multiplies<Scalar>());
		norm = thrust::reduce(thrust::device,temp.begin(),temp.end());
		norm = std::sqrt(norm);
	}
	x.setvalues({xx.begin(),xx.end()});
}