#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/copy.h>
#include<thrust/sort.h>
#include<thrust/execution_policy.h>

#include<vector>
#include "BICGSTAB.cuh"

CudaVector::~CudaVector()
{
	hipFree(values);
}

void CudaVector::AllocateData(const Vector& vec)
{
	hipMalloc((void**)&values, sizeof(double) * row);
	hipMemcpy(values, vec.generateScalar().data(), sizeof(double) * vec.size(),hipMemcpyHostToDevice);
}

CudaSPVector::~CudaSPVector()
{
	hipFree(vec);
}

void CudaSPVector::AllocateData(const Vector& vector)
{
	std::vector<IndexValue> vv;
	for (int i = 0; i < vector.size(); i++)
	{
		if(vector[i] != 0)
			vv.push_back({i,vector[i]});
	}
	
	hipMalloc((void**)&vec, sizeof(IndexValue) * vv.size());
	hipMemcpy(vec, vv.data(), sizeof(IndexValue) * vv.size(),hipMemcpyHostToDevice);
}

CudaSPMatrix::~CudaSPMatrix()
{
	for (int i = 0; i < row; i++)
	{
		hipFree(matrix[i]);
	}
	hipFree(dev_matrix);
}

void CudaSPMatrix::AllocateData(const SymetrixSparseMatrix& mat)
{
	matrix = new IndexValue*[row];
	for (int i = 0; i < row; i++)
	{
		std::vector<IndexValue> vv;
		for (auto& kv:mat.getRow(i))
		{
			if(kv.second != 0 && kv.first >= row)
				vv.push_back({(int)kv.first,kv.second});
		}
		hipMalloc((void**)&matrix[i], sizeof(IndexValue) * vv.size());
		hipMemcpy(matrix[i], vv.data(), sizeof(IndexValue) * vv.size(),hipMemcpyHostToDevice);
	}
	
	hipMalloc((void**)&dev_matrix, sizeof(IndexValue*) * row);
	hipMemcpy(dev_matrix, matrix, sizeof(IndexValue*) * row,hipMemcpyHostToDevice);
}

__global__ void compute(int* a,int* b,int length)
{
	auto id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= length)
		return;
	// b[id] = a[id] + 1;
	// b[id] = 1;
	printf("asdas");
}

void BiCGSTAB(const SymetrixSparseMatrix& A,Vector& x,const Vector& b)
{
	
	dim3 blockSize(32 ,32);
	dim3 threadSize(32, 32);
	int* a,*b2;
	const int length = 4;
	hipMalloc((void**)&a, sizeof(int) * length);
	hipMalloc((void**)&b2, sizeof(int) * length);
	
	compute << <blockSize, threadSize >> > (a,b2,length);
	int bb[length];

	hipMemcpy(bb, b2, sizeof(int) * length,hipMemcpyDeviceToHost);
	hipFree(a);
	hipFree(b2);
}